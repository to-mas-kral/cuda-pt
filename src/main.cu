#include <bit>
#include <chrono>

#include <CLI/CLI.hpp>
#include <fmt/core.h>
#include <optix.h>
#include <optix_function_table_definition.h>
#include <optix_host.h>
#include <optix_stubs.h>
#include <spdlog/spdlog.h>

#include "io/progress_bar.h"
#include "kernels/megakernel.h"
#include "kernels/raygen.h"
#include "optix_as.h"
#include "optix_common.h"
#include "optix_renderer.h"
#include "render_context_common.h"
#include "scene_loader.h"
#include "utils/cuda_err.h"
#include "utils/image_writer.h"
#include "utils/shared_vector.h"

int main(int argc, char **argv) {
    auto optix_context = init_optix();

    // TODO: wrap this in some class... need to have a block so that OptixRenderer
    // destructor is called before resetting the device at the end of main()...
    {
        /*
         * Parse comdline arguments
         * */

        u32 num_samples = 32;
        bool silent = false;
        bool optix = true;
        std::string scene_path{};

        CLI::App app{"A CUDA path-tracer project for PGRF3 by Tomáš Král, 2023."};
        // argv = app.ensure_utf8(argv);

        app.add_option("--samples", num_samples, "Number of samples.");
        app.add_option("-s,--scene", scene_path, "Path to the scene file.");
        app.add_flag("--silent,!--no-silent", silent, "Silent run.")->default_val(true);
        app.add_flag("--optix,!--no-optix", optix, "Use OptiX.")->default_val(true);

        CLI11_PARSE(app, argc, argv);

        if (silent) {
            spdlog::set_level(spdlog::level::err);
        }

        /*
         * Load scene attribs from the scene file
         * */

        SceneLoader scene_loader;
        try {
            scene_loader = SceneLoader(scene_path);
        } catch (const std::exception &e) {
            spdlog::error("Error while parsing the scene file");
            return 1;
        };
        auto attrib_result = scene_loader.load_scene_attribs();
        if (!attrib_result.has_value()) {
            spdlog::error("Error while getting scene attribs");
            return 1;
        }
        SceneAttribs attribs = attrib_result.value();

        /*
         * Set up render context
         * */

        // TODO: could probably make some template class for this...
        RenderContext *rc;
        CUDA_CHECK(hipMallocManaged((void **)&rc, sizeof(RenderContext)));
        auto rcx = new (rc) RenderContext(num_samples, attribs);

        /*
         * Load the scene
         * */

        spdlog::info("Loading the scene");
        try {
            scene_loader.load_scene(rc);
        } catch (const std::exception &e) {
            spdlog::error("Error while loading the scene");
            return 1;
        }
        spdlog::info("Scene loaded");

        rc->fixup_geometry_pointers();

        spdlog::info("Creating OptiX acceleration structure");
        auto optix_as = OptixAS(rc, optix_context);
        spdlog::info("OptiX acceleration structure initialized");
        auto optix_renderer = OptixRenderer(rc, optix_context, &optix_as);

        /*
         * Start rendering
         * */

        dim3 blocks_dim = rc->get_blocks_dim();
        dim3 threads_dim = rc->get_threads_dim();

        spdlog::info("Rendering a {}x{} image at {} samples.", attribs.resx, attribs.resy,
                     num_samples);

        if (!optix) {
            spdlog::info("Creating BVH acceleration structure");
            rc->make_acceleration_structure();
            spdlog::info("BVH acceleration structure created");

            spdlog::info("Pixel grid split into {} blocks with {} threads each.",
                         blocks_dim.x * blocks_dim.y, threads_dim.x * threads_dim.y);
        }

        PtParams params{};
        // Pass straight to params due to performance reasons...
        // No need to traverse 1 extra pointer...
        params.rc = rc;
        params.fb = &rc->get_framebuffer();
        params.meshes = rc->get_meshes().get_ptr();
        params.materials = rc->get_materials().get_ptr();
        params.lights = rc->get_lights().get_ptr();
        params.textures = rc->get_textures().get_ptr();

        ProgressBar pb;

        const auto start{std::chrono::steady_clock::now()};

        // OptiX path-tracer
        for (u32 s = 1; s <= num_samples; s++) {
            if (optix) {
                optix_renderer.launch(params, attribs.resx, attribs.resy);
            } else {
                render_megakernel<<<blocks_dim, threads_dim>>>(rc);

                hipDeviceSynchronize();
                CUDA_CHECK_LAST_ERROR();
            }

            const auto end{std::chrono::steady_clock::now()};
            const std::chrono::duration<f64> elapsed{end - start};

            // Update the framebuffer when the number of samples doubles...
            if (std::popcount(s) == 1) {

                ImageWriter::write_framebuffer("ptout.exr", rc->get_framebuffer(), s);
            }

            pb.print(s, num_samples, elapsed);
        }

        // spdlog::info("Shot a total of {} rays", rc->ray_counter.fetch_add(0));

        /*
         * Clean up and exit
         * */

        hipDeviceSynchronize();

        // Call the destructor manually, so the memory inside of RenderContext
        // deallocates.
        rc->~RenderContext();
        CUDA_CHECK(hipFree(rc));
        CUDA_CHECK_LAST_ERROR();
    }

    OPTIX_CHECK(optixDeviceContextDestroy(optix_context));
    CUDA_CHECK(hipDeviceReset());

    return 0;
}
