#include <bit>
#include <chrono>

#include <CLI/CLI.hpp>
#include <fmt/core.h>
#include <optix.h>
#include <optix_function_table_definition.h>
#include <optix_host.h>
#include <optix_stubs.h>
#include <spdlog/spdlog.h>

#include "io/image_writer.h"
#include "io/progress_bar.h"
#include "io/window.h"
#include "kernels/megakernel.h"
#include "kernels/raygen.h"
#include "optix_as.h"
#include "optix_common.h"
#include "optix_renderer.h"
#include "render_context_common.h"
#include "scene_loader.h"
#include "utils/cuda_err.h"
#include "utils/shared_vector.h"

// FIXME: there is a memory error in OptiX sphere acceleration creation, but seems to be
// an issue in Nvidia's code. Try when new CUDA version is released...
//==51801== Conditional jump or move depends on uninitialised value(s)
//==51801==    at 0x261576F9: ??? (in /usr/lib/libnvidia-rtcore.so.545.29.02)
//==51801==    by 0x2613C96D: ??? (in /usr/lib/libnvidia-rtcore.so.545.29.02)
//==51801==    by 0x2614B415: ??? (in /usr/lib/libnvidia-rtcore.so.545.29.02)
//==51801==    by 0x25F3B18D: ??? (in /usr/lib/libnvidia-rtcore.so.545.29.02)
//==51801==    by 0x25F3CCAA: ??? (in /usr/lib/libnvidia-rtcore.so.545.29.02)
//==51801==    by 0x225FA5CB: ??? (in /usr/lib/libnvoptix.so.545.29.02)
//==51801==    by 0x225F82DE: ??? (in /usr/lib/libnvoptix.so.545.29.02)
//==51801==    by 0x128B3F: optixAccelComputeMemoryUsage (optix_stubs.h:489)
//==51801==    by 0x12BFF6: OptixAS::create_as(OptixDeviceContext_t*,
//std::vector<OptixBuildInput, std::allocator<OptixBuildInput> > const&, unsigned long
//long*) (optix_as.h:83)
//==51801==    by 0x12CBC8: OptixAS::OptixAS(RenderContext*, OptixDeviceContext_t*)
//(optix_as.h:175)
//==51801==    by 0x125595: main (main.cu:104)
//==51801==  Uninitialised value was created by a stack allocation
//==51801==    at 0x2614B16B: ??? (in /usr/lib/libnvidia-rtcore.so.545.29.02)

int main(int argc, char **argv) {
    auto optix_context = init_optix();

    // TODO: wrap this in some class... need to have a block so that OptixRenderer
    // destructor is called before resetting the device at the end of main()...
    {
        /*
         * Parse comdline arguments
         * */

        u32 num_samples = 32;
        bool silent = false;
        bool optix = true;
        std::string scene_path{};

        CLI::App app{"A CUDA path-tracer project for PGRF3 by Tomáš Král, 2023."};
        // argv = app.ensure_utf8(argv);

        app.add_option("--samples", num_samples, "Number of samples.");
        app.add_option("-s,--scene", scene_path, "Path to the scene file.");
        app.add_flag("--silent,!--no-silent", silent, "Silent run.")->default_val(true);
        app.add_flag("--optix,!--no-optix", optix, "Use OptiX.")->default_val(true);

        CLI11_PARSE(app, argc, argv);

        spdlog::set_level(spdlog::level::info);

        if (silent) {
            spdlog::set_level(spdlog::level::err);
        }

        /*
         * Load scene attribs from the scene file
         * */

        SceneLoader scene_loader;
        try {
            scene_loader = SceneLoader(scene_path);
        } catch (const std::exception &e) {
            spdlog::error("Error while parsing the scene file");
            return 1;
        };
        auto attrib_result = scene_loader.load_scene_attribs();
        if (!attrib_result.has_value()) {
            spdlog::error("Error while getting scene attribs");
            return 1;
        }
        SceneAttribs attribs = attrib_result.value();

        /*
         * Window setup
         * */
        auto window = Window(attribs.resx, attribs.resy);

        /*
         * Set up render context
         * */

        // TODO: could probably make some template class for this...
        RenderContext *rc;
        CUDA_CHECK(hipMallocManaged((void **)&rc, sizeof(RenderContext)));
        auto rcx = new (rc) RenderContext(num_samples, attribs);

        /*
         * Load the scene
         * */

        spdlog::info("Loading the scene");
        try {
            scene_loader.load_scene(rc);
        } catch (const std::exception &e) {
            spdlog::error("Error while loading the scene {}", e.what());
            return 1;
        }
        spdlog::info("Scene loaded");

        rc->geometry.fixup_geometry_pointers();

        spdlog::info("Creating OptiX acceleration structure");
        auto optix_as = OptixAS(rc, optix_context);
        spdlog::info("OptiX acceleration structure initialized");
        auto optix_renderer = OptixRenderer(rc, optix_context, &optix_as);

        /*
         * Start rendering
         * */

        dim3 blocks_dim = rc->blocks_dim;
        dim3 threads_dim = rc->THREADS_DIM;

        spdlog::info("Rendering a {}x{} image at {} samples.", attribs.resx, attribs.resy,
                     num_samples);

        if (!optix) {
            spdlog::info("Creating BVH acceleration structure");
            rc->make_acceleration_structure();
            spdlog::info("BVH acceleration structure created");

            spdlog::info("Pixel grid split into {} blocks with {} threads each.",
                         blocks_dim.x * blocks_dim.y, threads_dim.x * threads_dim.y);
        }

        PtParams params{};
        // Pass straight to params due to performance reasons...
        // No need to traverse 1 extra pointer...
        params.rc = rc;
        params.fb = &rc->fb;
        params.meshes = rc->geometry.meshes.meshes.get_ptr();
        params.materials = rc->materials.get_ptr();
        params.lights = rc->lights.get_ptr();
        params.textures = rc->textures.get_ptr();

        ProgressBar pb;

        const auto start{std::chrono::steady_clock::now()};

        // OptiX path-tracer
        for (u32 s = 1; s <= num_samples; s++) {
            params.sample_index = s - 1;
            if (optix) {
                optix_renderer.launch(params, attribs.resx, attribs.resy);
            } else {
                render_megakernel<<<blocks_dim, threads_dim>>>(rc);

                hipDeviceSynchronize();
                CUDA_CHECK_LAST_ERROR();
            }

            const auto end{std::chrono::steady_clock::now()};
            const std::chrono::duration<f64> elapsed{end - start};

            // Update the framebuffer when the number of samples doubles...
            if (std::popcount(s) == 1) {
                window.update(rc->fb, s);
                ImageWriter::write_framebuffer("ptout.exr", rc->fb, s);
            }

            pb.print(s, num_samples, elapsed);
        }

        // spdlog::info("Shot a total of {} rays", rc->ray_counter.fetch_add(0));

        /*
         * Clean up and exit
         * */

        window.close();

        hipDeviceSynchronize();

        // Call the destructor manually, so the memory inside of RenderContext
        // deallocates.
        rc->~RenderContext();
        CUDA_CHECK(hipFree(rc));
        CUDA_CHECK_LAST_ERROR();
    }

    OPTIX_CHECK(optixDeviceContextDestroy(optix_context));
    CUDA_CHECK(hipDeviceReset());

    return 0;
}
