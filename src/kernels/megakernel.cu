#include "hip/hip_runtime.h"

#include "megakernel.h"

#include "../geometry/intersection.h"
#include "../integrator/utils.h"
#include "../render_context_common.h"
#include "../utils/rng.h"
#include "raygen.h"

// (s, t) are coords in screen space
__device__ vec3 render(RenderContext *rc, u32 x, u32 y) {
    auto pixel_index = rc->get_framebuffer().pixel_index(x, y);
    hiprandState *rand_state = &rc->get_framebuffer().get_rand_state()[pixel_index];

    Ray ray = gen_ray(x, y, &rc->get_framebuffer(), rc);

    /*
     * Iterative naive path tracing
     * */
    u32 depth = 1;
    vec3 throughput = vec3(1.f);
    vec3 radiance = vec3(0.f);

    while (true) {
        // rc->ray_counter.fetch_add(1);
        auto pot_its = rc->intersect_scene(ray);
        if (pot_its.has_value()) {
            auto its = pot_its.value();
            auto material = &rc->get_materials()[its.mesh->material_id];

            vec3 emission = vec3(0.f);
            if (its.mesh->light_id.has_value()) {
                auto light_id = its.mesh->light_id.value();
                emission = rc->get_lights()[light_id].emission();
            }

            if (glm::dot(-ray.dir, its.normal) < 0.f) {
                its.normal = -its.normal;
                emission = vec3(0.f);
            }

            vec3 sample_dir = material->sample(its.normal, -ray.dir, rand_state);
            // TODO: what to do when cos_theta is 0 ? this minimum value is a band-aid
            // solution...
            f32 cos_theta = max(glm::dot(its.normal, sample_dir), 0.0001f);

            f32 pdf = material->pdf(cos_theta);
            // FIXME: megakernel textures
            vec3 brdf = material->eval(material, rc->get_textures().get_ptr(), vec2(0.));

            radiance += throughput * emission;
            throughput *= brdf * cos_theta * (1.f / pdf);

            auto rr = russian_roulette(depth, rand_state, throughput);

            if (!rr.has_value()) {
                return radiance;
            } else {
                auto roulette_compensation = rr.value();
                throughput *= 1.f / roulette_compensation;
            }

            Ray new_ray = spawn_ray(its, sample_dir);
            ray = new_ray;
            depth++;
        } else {
            // Ray has escaped the scene
            if (!rc->has_envmap) {
                return vec3(0.);
            } else {
                const Envmap *envmap = rc->get_envmap();
                vec3 envrad = envmap->sample(ray);
                radiance += throughput * envrad;
                return radiance;
            }
        }
    }
}

/// The "megakernel" approach to path-tracing on the GPU
__global__ void render_megakernel(RenderContext *rc) {
    u32 pixel_index = rc->get_framebuffer().pixel_index(blockDim, blockIdx, threadIdx);

    if (pixel_index < rc->get_framebuffer().num_pixels()) {
        auto [x, y] = rc->get_framebuffer().pixel_coords(blockDim, blockIdx, threadIdx);

        vec3 radiance = render(rc, x, y);
        rc->get_framebuffer().get_pixels()[pixel_index] += radiance;
    }
}
