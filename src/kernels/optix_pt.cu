#include "hip/hip_runtime.h"
#include "optix_pt.h"

#include <hip/hip_runtime.h>
#include <glm/gtx/norm.hpp>
#include <optix.h>
#include <optix_device.h>

#include "../integrator/utils.h"
#include "../utils/numtypes.h"
#include "../utils/sampler.h"
#include "raygen.h"

extern "C" {
__constant__ PtParams params;
}

const u32 NO_HIT = 0;
const u32 HIT_TRIANGLE = 1;
const u32 HIT_SPHERE = 2;

static __forceinline__ __device__ void set_payload_miss() { optixSetPayload_0(NO_HIT); }

static __forceinline__ __device__ void
set_payload_hit_triangle(u32 prim_index, u32 mesh_id, float2 barycentrics,
                         hipDeviceptr_t pos, hipDeviceptr_t indices, hipDeviceptr_t normals,
                         hipDeviceptr_t uvs) {
    // TODO: could pack hit/miss into the prim_index...
    optixSetPayload_0(HIT_TRIANGLE);
    optixSetPayload_1(prim_index);
    optixSetPayload_2(mesh_id);

    optixSetPayload_3(__float_as_uint(barycentrics.x));
    optixSetPayload_4(__float_as_uint(barycentrics.y));

    optixSetPayload_5(static_cast<u32>(pos));
    optixSetPayload_6((static_cast<u64>(pos) & 0xFFFF'FFFF'0000'0000U) >> 32U);
    optixSetPayload_7(static_cast<u32>(indices));
    optixSetPayload_8((static_cast<u64>(indices) & 0xFFFF'FFFF'0000'0000U) >> 32U);
    optixSetPayload_9(static_cast<u32>(normals));
    optixSetPayload_10((static_cast<u64>(normals) & 0xFFFF'FFFF'0000'0000U) >> 32U);
    optixSetPayload_11(static_cast<u32>(uvs));
    optixSetPayload_12((static_cast<u64>(uvs) & 0xFFFF'FFFF'0000'0000U) >> 32U);
}

static __forceinline__ __device__ void set_payload_hit_sphere(u32 prim_index,
                                                              u32 material_id,
                                                              u32 light_id,
                                                              bool has_light, f32 t) {
    optixSetPayload_0(HIT_SPHERE);
    optixSetPayload_1(prim_index);
    optixSetPayload_2(material_id);
    optixSetPayload_3(light_id);
    optixSetPayload_4(has_light);
    optixSetPayload_5(__float_as_uint(t));
}

static __forceinline__ __device__ Intersection
get_triangle_its(u32 bar_y, u32 bar_z, u32 triangle_index, u32 mesh_id, hipDeviceptr_t d_pos,
                 hipDeviceptr_t d_indices, hipDeviceptr_t d_normals, hipDeviceptr_t d_uvs) {

    f32 bar_y_f = __uint_as_float(bar_y);
    f32 bar_z_f = __uint_as_float(bar_z);
    vec3 bar = vec3(1.f - bar_y_f - bar_z_f, bar_y_f, bar_z_f);

    auto mesh_o = &params.meshes[mesh_id];

    vec3 *positions = (vec3 *)d_pos;
    u32 *indices = (u32 *)d_indices;
    vec3 *normals = (vec3 *)d_normals;
    vec2 *uvs = (vec2 *)d_uvs;

    u32 i0 = indices[3 * triangle_index];
    u32 i1 = indices[3 * triangle_index + 1];
    u32 i2 = indices[3 * triangle_index + 2];

    vec3 p0 = positions[i0];
    vec3 p1 = positions[i1];
    vec3 p2 = positions[i2];

    vec3 pos = barycentric_interp(bar, p0, p1, p2);

    vec3 normal =
        Meshes::calc_normal(mesh_o->has_normals, i0, i1, i2, normals, bar, p0, p1, p2);
    vec2 uv = Meshes::calc_uvs(mesh_o->has_uvs, i0, i1, i2, uvs, bar);

    return Intersection{
        .material_id = mesh_o->material_id,
        .light_id = mesh_o->lights_start_id + triangle_index,
        .has_light = mesh_o->has_light,
        .normal = normal,
        .pos = pos,
        .uv = uv,
    };
}

__device__ __forceinline__ hipDeviceptr_t unpack_ptr(u32 hi, u32 lo) {
    return (static_cast<u64>(hi) << 32U) | static_cast<u64>(lo);
}

__device__ __forceinline__ Intersection get_sphere_its(u32 sphere_index,
                                                         u32 material_id, u32 light_id,
                                                         u32 has_light, Spheres &spheres,
                                                         const vec3 &pos) {
    vec3 center = spheres.centers[sphere_index];

    vec3 normal = Spheres::calc_normal(pos, center);
    vec2 uv = Spheres::calc_uvs(normal);

    return Intersection{
        .material_id = material_id,
        .light_id = light_id,
        .has_light = bool(has_light),
        .normal = normal,
        .pos = pos,
        .uv = uv,
    };
}

__device__ __forceinline__ Intersection get_its(Scene *sc, u32 p1, u32 p2, u32 p3,
                                                  u32 p4, u32 p5, u32 p6, u32 p7, u32 p8,
                                                  u32 p9, u32 p10, u32 p11, u32 p12,
                                                  u32 did_hit, Ray &ray) {
    if (did_hit == HIT_TRIANGLE) {
        u32 prim_index = p1;
        u32 mesh_id = p2;
        u32 bar_y = p3;
        u32 bar_z = p4;
        u32 pos_lo = p5;
        u32 pos_hi = p6;
        u32 indices_lo = p7;
        u32 indices_hi = p8;
        u32 normals_lo = p9;
        u32 normals_hi = p10;
        u32 uvs_lo = p11;
        u32 uvs_hi = p12;

        hipDeviceptr_t d_pos = unpack_ptr(pos_hi, pos_lo);
        hipDeviceptr_t d_indices = unpack_ptr(indices_hi, indices_lo);
        hipDeviceptr_t d_normals = unpack_ptr(normals_hi, normals_lo);
        hipDeviceptr_t d_uvs = unpack_ptr(uvs_hi, uvs_lo);

        return get_triangle_its(bar_y, bar_z, prim_index, mesh_id, d_pos, d_indices,
                                d_normals, d_uvs);
    } else {
        // Sphere
        u32 sphere_index = p1;
        u32 material_id = p2;
        u32 light_id = p3;
        u32 has_light = p4;
        f32 t = __uint_as_float(p5);

        Spheres &spheres = sc->geometry.spheres;
        vec3 pos = ray.o + ray.dir * t;

        return get_sphere_its(sphere_index, material_id, light_id, has_light, spheres,
                              pos);
    }
}

// Multisple Importance Sampling for lights
__device__ __forceinline__ void
light_mis(const Intersection &its, const Ray &traced_ray, const Ray &bxdf_ray,
          const LightSample &light_sample, const ShapeSample &shape_sample,
          const Material *material, vec3 *radiance, const vec3 &throughput) {
    vec3 light_pos = shape_sample.pos;
    vec3 pl_norm = glm::normalize(light_pos - its.pos);
    f32 pl_mag_sq = glm::length2(light_pos - its.pos);
    f32 cos_light = max(glm::dot(shape_sample.normal, -pl_norm), 0.000001f);

    auto sgeom_light = get_shading_geom(its.normal, pl_norm, -traced_ray.dir);

    // Quickly precheck if light is reachable
    if (sgeom_light.cos_theta > 0.f && cos_light > 0.f) {
        f32 pl_mag = glm::length(light_pos - its.pos);

        // Use the origin of the BXDF ray, which is already offset from the surface so
        // that it doesn't self-intersect.
        vec3 lrd = glm::normalize(light_pos - bxdf_ray.o);
        float3 raydir = make_float3(lrd.x, lrd.y, lrd.z);
        float3 rayorig = make_float3(bxdf_ray.o.x, bxdf_ray.o.y, bxdf_ray.o.z);
        u32 did_hit = 1;
        // TODO: didn!t get much of a speedup, investigate
        // https://www.willusher.io/graphics/2019/09/06/faster-shadow-rays-on-rtx
        optixTrace(params.gas_handle, rayorig, raydir, 0.0001f, pl_mag - 0.001f, 0.0f,
                   OptixVisibilityMask(255),
                   OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT |
                       OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
                   0, 1, 0, did_hit);

        if (!did_hit) {
            // Probability of sampling this light in terms of solid angle from the
            // probability distribution of the lights. Formula from
            // https://www.pbr-book.org/4ed/Radiometry,_Spectra,_and_Color/Working_with_Radiometric_Integrals#IntegralsoverArea
            f32 pdf_light = shape_sample.pdf * light_sample.pdf * (pl_mag_sq / cos_light);

            vec3 bxdf_light = material->eval(params.textures, its.uv);
            f32 weight_light = mis_power_heuristic(pdf_light, material->pdf(sgeom_light));

            vec3 light_emission = light_sample.light.emitter.emission();

            *radiance += bxdf_light * sgeom_light.cos_theta * (1.f / pdf_light) *
                         light_emission * weight_light * throughput;
        }
    }
}

extern "C" __global__ void __raygen__rg() {
    const uint3 pixel = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    auto rc = params.rc;
    auto sc = &rc->scene;
    auto pixel_index = ((dim.y - 1U - pixel.y) * dim.x) + pixel.x;

    auto sampler = &params.fb->get_rand_state()[pixel_index];

    auto cam_sample = vec2(sampler->sample(), sampler->sample());

    auto ray = gen_ray(pixel.x, pixel.y, dim.x, dim.y, cam_sample, rc);

    u32 depth = 1;
    vec3 throughput = vec3(1.f);
    vec3 radiance = vec3(0.f);

    vec3 last_hit_pos = vec3(0.f);
    f32 last_pdf_bxdf = 0.f;

    while (true) {
        float3 raydir = make_float3(ray.dir.x, ray.dir.y, ray.dir.z);
        float3 rayorig = make_float3(ray.o.x, ray.o.y, ray.o.z);

        u32 p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12;
        optixTrace(params.gas_handle, rayorig, raydir, 0.0f, 1e16f, 0.0f,
                   OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT, 0, 1, 0, p0,
                   p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12);

        u32 did_hit = p0;
        if (did_hit) {
            auto bsdf_sample = vec2(sampler->sample(), sampler->sample());
            auto rr_sample = sampler->sample();
            Intersection its = get_its(sc, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11,
                                         p12, did_hit, ray);

            auto material = &params.materials[its.material_id];
            bool is_frontfacing = glm::dot(-ray.dir, its.normal) > 0.f;

            if (its.has_light && is_frontfacing) {
                vec3 emission = params.lights[its.light_id].emitter.emission();

                if (depth == 1) {
                    // Primary ray hit, can apply MIS...
                    radiance += throughput * emission;
                } else {
                    vec3 pl_norm = glm::normalize(its.pos - last_hit_pos);
                    f32 pl_mag_sq = glm::length2(its.pos - last_hit_pos);
                    f32 cos_light = glm::dot(its.normal, -pl_norm);

                    // last_pdf_bxdf is the probability of this light having been sampled
                    // from the probability distribution of the BXDF of the *preceding*
                    // hit.

                    // TODO: currently calculating the shape PDF by assuming pdf = 1. /
                    // area will have to change with non-uniform sampling !
                    f32 light_area =
                        sc->geometry.shape_area(sc->lights[its.light_id].shape);

                    // pdf_light is the probability of this light being sampled from the
                    // probability distribution of the lights.
                    f32 pdf_light = sc->light_sampler.light_sample_pdf(its.light_id) *
                                    pl_mag_sq / (light_area * cos_light);

                    f32 bxdf_weight = mis_power_heuristic(last_pdf_bxdf, pdf_light);
                    radiance += throughput * bxdf_weight * emission;
                }
            }

            if (!is_frontfacing) {
                its.normal = -its.normal;
            }

            vec3 sample_dir = material->sample(its.normal, -ray.dir, bsdf_sample);
            auto sgeom_bxdf = get_shading_geom(its.normal, sample_dir, -ray.dir);

            Ray bxdf_ray = spawn_ray(its, sample_dir);

            f32 pdf = material->pdf(sgeom_bxdf);
            vec3 bxdf = material->eval(params.textures, its.uv);

            f32 light_sample = sampler->sample();
            auto sampled_light = sc->sample_lights(light_sample);
            if (sampled_light.has_value()) {
                // TODO: create a template for creating these vector samples...
                vec3 shape_rng =
                    vec3(sampler->sample(), sampler->sample(), sampler->sample());
                auto shape_sample = sc->geometry.sample_shape(
                    sampled_light.value().light.shape, its.pos, shape_rng);

                light_mis(its, ray, bxdf_ray, sampled_light.value(), shape_sample,
                          material, &radiance, throughput);
            }

            auto rr = russian_roulette(depth, rr_sample, throughput);
            if (!rr.has_value()) {
                break;
            }

            auto roulette_compensation = rr.value();

            throughput *=
                bxdf * sgeom_bxdf.cos_theta * (1.f / (pdf * roulette_compensation));

            ray = bxdf_ray;
            last_hit_pos = its.pos;
            last_pdf_bxdf = pdf;
            depth++;
        } else {
            // TODO: move into miss program to reduce divergence ?
            // Ray has escaped the scene
            if (!sc->has_envmap) {
                break;
            } else {
                const Envmap *envmap = &sc->envmap;
                vec3 envrad = envmap->sample(ray);
                radiance += throughput * envrad;
                break;
            }
        }
    }

    params.fb->get_pixels()[pixel_index] += radiance;
}

extern "C" __global__ void __miss__ms() { set_payload_miss(); }

/*
 * From Nvidia docs:
 * "It is generally more efficient to have one hit shader handle multiple primitive
 * types (by switching on the value of optixGetPrimitiveType), rather than have several
 * hit shaders that implement the same ray behavior but differ only in the type of
 * geometry they expect."
 * */
extern "C" __global__ void __closesthit__ch() {
    auto *hit_data = reinterpret_cast<PtHitGroupData *>(optixGetSbtDataPointer());

    auto type = optixGetPrimitiveType();

    if (type == OPTIX_PRIMITIVE_TYPE_TRIANGLE) {
        // Triangles
        const float2 barycentrics = optixGetTriangleBarycentrics();
        const u32 prim_index = optixGetPrimitiveIndex();

        set_payload_hit_triangle(prim_index, hit_data->mesh.mesh_id, barycentrics,
                                 hit_data->mesh.pos, hit_data->mesh.indices,
                                 hit_data->mesh.normals, hit_data->mesh.uvs);
    } else {
        // Spheres
        // TODO: maybe add sphere_id to hit_data... would be safer if multiple
        // sphere GASes are used in the future...
        const u32 sphere_index = optixGetSbtGASIndex();
        f32 t = optixGetRayTmax();

        set_payload_hit_sphere(sphere_index, hit_data->sphere.material_id,
                               hit_data->sphere.light_id, hit_data->sphere.has_light, t);
    }
}
