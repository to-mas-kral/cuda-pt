#include "hip/hip_runtime.h"
#include "optix_pt.h"

#include <hip/hip_runtime.h>
#include <optix.h>
#include <optix_device.h>

#include "../geometry/intersection.h"
#include "../integrator/utils.h"
#include "../utils/numtypes.h"
#include "raygen.h"

extern "C" {
__constant__ PtParams params;
}

// TODO: could pack hit/miss into the prim_index...

// Payload structure
// 0 - hit or miss
// 1 - prim_index
// 2 - mesh_index
// 3 - barycentric x
// 4 - barycentric y
// 5, 6 - pos pointer
// 7, 8 - indices pointer

static __forceinline__ __device__ void set_payload_miss() { optixSetPayload_0(0); }

static __forceinline__ __device__ void set_payload_hit(u32 prim_index, u32 mesh_id,
                                                       float2 barycentrics,
                                                       hipDeviceptr_t pos,
                                                       hipDeviceptr_t indices) {
    optixSetPayload_0(1);
    optixSetPayload_1(prim_index);
    optixSetPayload_2(mesh_id);

    optixSetPayload_3(__float_as_uint(barycentrics.x));
    optixSetPayload_4(__float_as_uint(barycentrics.y));

    optixSetPayload_5(static_cast<u32>(pos));
    optixSetPayload_6((static_cast<u64>(pos) & 0xFFFF'FFFF'0000'0000U) >> 32U);
    optixSetPayload_7(static_cast<u32>(indices));
    optixSetPayload_8((static_cast<u64>(indices) & 0xFFFF'FFFF'0000'0000U) >> 32U);
}

extern "C" __global__ void __raygen__rg() {
    const uint3 pixel = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    auto rc = params.rc;
    auto pixel_index = params.fb->pixel_index(pixel.x, pixel.y);
    hiprandState *rand_state = &params.fb->get_rand_state()[pixel_index];

    auto ray = gen_ray(pixel.x, pixel.y, params.fb, rc);

    u32 depth = 1;
    vec3 throughput = vec3(1.f);
    vec3 radiance = vec3(0.f);

    while (true) {
        float3 raydir = make_float3(ray.dir.x, ray.dir.y, ray.dir.z);
        float3 rayorig = make_float3(ray.o.x, ray.o.y, ray.o.z);

        u32 did_hit = 0xdeadbeef, prim_index = 0xdeadbeef, mesh_id = 0xdeadbeef,
            bar_y = 0xdeadbeef, bar_z = 0xdeadbeef, pos_lo = 0xdeadbeef,
            pos_hi = 0xdeadbeef, indices_lo = 0xdeadbeef, indices_hi = 0xdeadbeef;
        optixTrace(params.gas_handle, rayorig, raydir, 0.0f, 1e16f, 0.0f,
                   OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT, 0, 1, 0,
                   did_hit, prim_index, mesh_id, bar_y, bar_z, pos_lo, pos_hi, indices_lo,
                   indices_hi);

        hipDeviceptr_t d_pos = (static_cast<u64>(pos_hi) << 32U) | static_cast<u64>(pos_lo);
        hipDeviceptr_t d_indices =
            (static_cast<u64>(indices_hi) << 32U) | static_cast<u64>(indices_lo);

        if (did_hit) {
            f32 bar_y_f = __uint_as_float(bar_y);
            f32 bar_z_f = __uint_as_float(bar_z);
            vec3 bar = vec3(1.f - bar_y_f - bar_z_f, bar_y_f, bar_z_f);

            auto mesh = &params.meshes[mesh_id];
            auto material = &params.materials[mesh->material_id];

            vec3 *positions = (vec3 *)d_pos;
            u32 *indices = (u32 *)d_indices;

            u32 i0 = indices[3 * prim_index];
            u32 i1 = indices[3 * prim_index + 1];
            u32 i2 = indices[3 * prim_index + 2];

            vec3 p0 = positions[i0];
            vec3 p1 = positions[i1];
            vec3 p2 = positions[i2];

            vec3 pos = bar.x * p0 + bar.y * p1 + bar.z * p2;

            // TODO: adjust when mesh normals are added
            vec3 v0 = p1 - p0;
            vec3 v1 = p2 - p0;
            vec3 normal = glm::normalize(cross(v0, v1));
            if (glm::any(glm::isnan(normal))) {
                // Degenerate triangle...
                // TODO: HACK
                normal = glm::normalize(-ray.dir);
            }

            vec3 emission = vec3(0.f);
            if (mesh->has_light()) {
                emission = params.lights[mesh->light_id].emission();
            }

            if (glm::dot(-ray.dir, normal) < 0.f) {
                normal = -normal;
                emission = vec3(0.f);
            }

            Intersection its{
                .pos = pos,
                .normal = normal,
                .t = -1.f, // TODO: t used for anything ?
                .mesh = mesh,
            };

            vec3 sample_dir = material->sample(normal, -ray.dir, rand_state);
            // TODO: what to do when cos_theta is 0 ? this minimum value is a band-aid
            // solution...
            f32 cos_theta = max(glm::dot(normal, sample_dir), 0.0001f);

            f32 pdf = material->pdf(cos_theta);
            vec3 brdf = material->eval();

            radiance += throughput * emission;
            throughput *= brdf * cos_theta * (1.f / pdf);

            auto [should_terminate, roulette_compensation] =
                russian_roulette(depth, rand_state, throughput);

            if (should_terminate) {
                break;
            }

            throughput *= 1.f / roulette_compensation;

            Ray new_ray = spawn_ray(its, sample_dir);
            ray = new_ray;
            depth++;
        } else {
            // Ray has escaped the scene
            if (!rc->has_envmap) {
                radiance = vec3(0.f);
                break;
            } else {
                const Envmap *envmap = rc->get_envmap();
                vec3 envrad = envmap->sample(ray);
                radiance += throughput * envrad;
                break;
            }
        }
    }

    params.fb->get_pixels()[pixel_index] += radiance;
}

extern "C" __global__ void __miss__ms() { set_payload_miss(); }

extern "C" __global__ void __closesthit__ch() {
    PtHitGroupData *hit_data =
        reinterpret_cast<PtHitGroupData *>(optixGetSbtDataPointer());
    const float2 barycentrics = optixGetTriangleBarycentrics();
    const u32 prim_index = optixGetPrimitiveIndex();

    set_payload_hit(prim_index, hit_data->mesh_id, barycentrics, hit_data->pos,
                    hit_data->indices);
}
